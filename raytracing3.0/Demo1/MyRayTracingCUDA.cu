#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "time.h" 
#include "MyCUDARayTracing.h"
#include "StaticConstants.h"

//������ʹ�õ�λ����
/*
���ǻ���һ���ļ��裬��Ϊ������ܳ��Ⱥ��ܿ�����
ÿ��������Ŀ���趨Ϊ2
�������֪����ǰ����Ϊ(x,y)
(int(x/2),int(y/2))��ʾ��ǰ���ڵ�����
��һά����grid������Ϊ int(x/2)+int(y/2)*ROW
����������Ҫ����������Ϣ����ָ����ǰ����λ�õĴ��䷽��
	������ڸ��ӱ߽紦������Ҫ�ı�
	����ں��ݽ��㴦
		��һ����	����
		�ڶ�����	(int(x/2)-1)+int(y/2)*ROW
		��������	(int(x/2)-1)+(int(y/2)-1)*ROW
		��������	(int(x/2))-1+int(y/2)*ROW
	����ں���߽紦
		ָ���������� int(x/2)+(int(y/2)-1)*ROW
	���������߽紦
		ָ��������� (int(x/2))-1+int(y/2)*ROW


*/



__device__ double directionIntersect(double x1, double y1, double x2, double y2, double x3, double y3) {
	//(x3-x1,y3-y1)X(x2-x1,y2-y1)
	return (y2 - y1)*(x3 - x1) - (x2 - x1)*(y3 - y1);
}
__device__ double Mymin(double x1, double x2) {
	if (x1 < x2)
		return x1;
	else
		return x2;
}
__device__ double Mymax(double x1, double x2) {
	return x1 > x2 ? x1 : x2;
}
__device__ double calDistance(double x1, double y1, double x2, double y2) {
	return sqrt((x2 - x1)*(x2 - x1) + (y2 - y1)*(y2 - y1));
}
__device__ bool onSegment(double x1, double y1, double x2, double y2, double x3, double y3) {
	if (Mymin(x1, x2) <= x3 && x3 < Mymax(x1, x2) && Mymin(y1, y2) < y3&&y3 < Mymax(y1, y2)) {
		return true;
	}
	else {
		return false;
	}
}
__device__ void calculPointOfIntersection(double x1, double y1, double x2, double y2, double x3, double y3, double x4, double y4, double &x, double &y) {
	double a = y2 - y1;
	double b = x1 - x2;
	double c = y4 - y3;
	double d = x3 - x4;
	double e = (y2 - y1)*x1 - (x2 - x1)*y1;
	double f = (y4 - y3)*x3 - (x4 - x3)*y3;
	x = (e*d - b * f) / (a*d - b * c);
	y = (a*f - e * c) / (a*d - b * c);
}
__device__ bool segmentsIntersect(double x1, double y1, double x2, double y2, double x3, double y3, double x4, double y4) {
	double d1 = directionIntersect(x3, y3, x4, y4, x1, y1);
	double d2 = directionIntersect(x3, y3, x4, y4, x2, y2);
	double d3 = directionIntersect(x1, y1, x2, y2, x3, y3);
	double d4 = directionIntersect(x1, y1, x2, y2, x4, y4);
	if (((d1 > 0 && d2 < 0) || (d1 < 0 && d2>0)) && ((d3 > 0 && d4 < 0) || (d3 < 0 && d4>0)))
		return true;
	else if (d1 == 0 && onSegment(x3, y3, x4, y4, x1, y1)) {
		return true;
	}
	else if (d2 == 0 && onSegment(x3, y3, x4, y4, x2, y2)) {
		return true;
	}
	else if (d3 == 0 && onSegment(x1, y1, x2, y2, x3, y3)) {
		return true;
	}
	else if (d4 == 0 && onSegment(x1, y1, x2, y2, x4, y4)) {
		return true;
	}
	else
		return false;
}

__device__ void normalizeVector(double &x, double &y) {
	double norm = sqrt(x*x + y * y);
	x = x / norm;
	y = y / norm;
}
/*
vector is input
points is output
*/
__global__ void judgeIsTouched(MyVector *vector, Grids *grids, Points *points, Point *point, int N) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < N) {
		int row = vector[i].row;
		int col = vector[i].col;
		int oldRow = row;
		int oldCol = col;
		double x = vector[i].x;
		double y = vector[i].y;
		double vectorX = vector[i].vectorX;
		double vectorY = vector[i].vectorY;
		while (true) {
			oldRow = row;
			oldCol = col;
			if (row < 0 || col < 0 || row >= ROW || col >= COL)
				break;
			int indexC = row * COL + col;
			//���ȼ���������һ���ߵĽ��㹹��һ���߶�
			int currentX = col * LENGTH;
			int currentY = row * LENGTH;
			//��Ҫ�ж��Ƿ����
			double newX = x, newY = y;
			if (vectorX > 0 && vectorY > 0) {
				if ((currentX + LENGTH - x)*vectorY > (currentY + LENGTH - y)*vectorX) {
					//top
					if (vectorY != 0) {
						newX = x + (currentX + LENGTH - x)*vectorX / vectorY;
						newY = currentY + LENGTH;
					}
					else {
						newX = currentX + LENGTH;
					}
					row++;
				}
				else if ((currentX + LENGTH - x)*vectorY == (currentY + LENGTH - y)*vectorX) {
					newX = currentX + LENGTH;
					newY = currentY + LENGTH;
					row++;
					col++;
				}
				else {
					//right
					newX = currentX + LENGTH;
					newY = vectorY / vectorX * (currentX + 2 - x) + y;
					col++;
				}
			}
			else if (vectorX > 0 && vectorY < 0) {
				if (-vectorY * (currentX + LENGTH - x) < vectorX*(y - currentY)) {
					//right
					newY = y + (currentX + 2 - x)*vectorY / vectorX;
					newX = currentX + LENGTH;
					col++;
				}
				else if (-vectorY * (currentX + LENGTH - x) == vectorX * (y - currentY)) {
					newX = currentX + LENGTH;
					newY = currentY;
					col++;
					row--;
				}
				else {
					//bottom
					newX = x - vectorX / vectorY * (y - currentY);
					newY = currentY;
					row--;
				}
			}
			else if (vectorX < 0 && vectorY>0) {
				if (-vectorX * (currentY + LENGTH - y) < vectorY*(x - currentX)) {
					//top
					newX = x + (vectorX / vectorY * (currentY + LENGTH - y));
					newY = currentY + LENGTH;
					row++;
				}
				else if (-vectorX * (currentY + LENGTH - y) == vectorY * (x - currentX)) {
					newX = currentX;
					newY = currentY + LENGTH;
					row++;
					col--;
				}
				else {
					//left
					newY = y + (vectorY / vectorX * (currentX - x));
					newX = currentX;
					col--;
				}
			}
			else if (vectorX < 0 && vectorY < 0) {
				if (vectorY*(x - currentX) < vectorX*(y - currentY)) {
					//bottom
					newX = x - vectorX / vectorY * (y - currentY);
					newY = currentY;
					row--;
				}
				else if (vectorY*(x - currentX) == vectorX * (y - currentY)) {
					newX = currentX;
					newY = currentY;
					row--;
					col--;
				}
				else {
					//left
					newY = y - (x - currentX)*vectorY / vectorX;
					newX = currentX;
					col--;
				}
			}
			else if (vectorX == 0) {
				newY = vectorY > 0 ? currentY + LENGTH : currentY;
				row = vectorY > 0 ? row + 1 : row - 1;
			}
			else if (vectorY == 0) {
				newX = vectorX > 0 ? currentX + LENGTH : currentY;
				col = vectorX > 0 ? col + 1 : col - 1;
			}
			//points[i].point[0].x = newX;
			//points[i].point[0].y = newY;
			//grids->grids[0].isContainsgrids[indexC].grids->isContains

			if (grids->grids[indexC].isContains) {

				//contains edge
				double distanceM = std::sqrt(2.0)*LENGTH;
				bool isFind = false;
				double insertPointOutX = 0, insertPointOutY = 0;
				int edgeIndex = 0;
				for (int k = 0; k < grids->grids[indexC].N; k++) {
					//grids->grids[0].edges[k].xstart
					if (segmentsIntersect(x, y, newX, newY, grids->grids[indexC].edges[k].xstart, grids->grids[indexC].edges[k].ystart, grids->grids[indexC].edges[k].xend, grids->grids[indexC].edges[k].yend) == true) {
						double insertPointX = 0, insertPointY = 0;
						calculPointOfIntersection(x, y, newX, newY, grids->grids[indexC].edges[k].xstart, grids->grids[indexC].edges[k].ystart, grids->grids[indexC].edges[k].xend, grids->grids[indexC].edges[k].yend, insertPointX, insertPointY);
						if (currentX > insertPointX || insertPointOutX > currentX + LENGTH || currentY > insertPointY || insertPointY > currentY + LENGTH) {
							//������������������
							continue;
						}
						double tempDistance = calDistance(x, y, insertPointX, insertPointY);
						//��ô���õ�Ŀ���Ǵ��ڷ�����µ������Ȼ����������ڣ�������һ��ǽ���ϣ��ű�Ȼ�����ǽ���ཻ�ģ������Ҫ���˵��������
						if (abs(insertPointX - x) < 0.0001 && abs(insertPointY - y) < 0.0001)
							continue;
						if (tempDistance < distanceM) {
							isFind = true;
							distanceM = tempDistance;
							insertPointOutX = insertPointX;
							insertPointOutY = insertPointY;
							edgeIndex = k;
						}
					}
				}
				//�Ѿ���������ˣ�����н��㣬����Ҫ�������ߴ���ķ���
				if (isFind) {
					double currentVectorX = x - insertPointOutX;
					double currentVectorY = y - insertPointOutY;
					//Ҫ��������
					//���������ǶԳ�
					// \  |  /
					//  \ | /
					//   \|/
					//------------------
					//����ǽ��ķ�����
					//grids->grids[indexC].edges[k]
					double edgeVectorX = -grids->grids[indexC].edges[edgeIndex].vectorX;
					double edgeVectorY = -grids->grids[indexC].edges[edgeIndex].vectorY;
					if (edgeVectorX*currentVectorX + edgeVectorY * currentVectorY < 0) {
						edgeVectorX = -edgeVectorX;
						edgeVectorY = -edgeVectorY;
					}
					double temp = currentVectorX * edgeVectorX + currentVectorY * edgeVectorY;//uv
					double edgeProjectVectorX = temp * edgeVectorX;
					double edgeProjectVectorY = temp * edgeVectorY;
					//������ϣ��������ݣ�������д�����Points
					vectorX = 2 * edgeProjectVectorX - currentVectorX;
					vectorY = 2 * edgeProjectVectorY - currentVectorY;
					normalizeVector(vectorX, vectorY);
					x = insertPointOutX;
					y = insertPointOutY;
					points[i].point[points[i].N].x = x;
					points[i].point[points[i].N].y = y;
					points[i].N++;
					row = oldRow;
					col = oldCol;
					//printf("%d\n", points[i].N);
					if (points[i].N > 19)
						break;
					if (points[i].N > MAX_REFLECTION_TIMES)
						break;
				}
			}
			else {
				//��������
				x = newX;
				y = newY;
			}
			//printf("current position:%f,%f", x, y);
			//�����newX��newY�ǽ�����������꣬��ʱ��Ҫ�жϸ����߶�������еı��Ƿ��ཻ
			//break;
			//if (grids[indexC].grids->isContains) {
			//	
			//	for (int k = 0; k < grids[indexC].grids->N; i++) {
			//		//judge weather is intersecting
			//	}

			//}
			//else {


			//	//��Ҫ�ҵ��������ĸ���
			//	//update vectors
			//	//remain direction update x and y
			//}
		}
		//vector[i].
	}
}
/*
grid_input:�洢��������Ϣ��ÿ�������д��ڱߣ���Щ��Ҳ�ṹ������
N:��ʾ�����������ȼ�����֣����N=360����ÿ���Ƕ�Ϊ1��
*/
vector<vector<double>> initCUDAInput(Grids *grids, double TX_X, double TX_Y, double RX_X, double RX_Y, int N) {
	//��Ҫ��ʼ������
	//grids->�洢����
	//������������Ϊ50*50��С������ɣ�������������С����Щ��������ȫ����Ϊ��
	//int N = 180;
	printf("Start...%d\n", sizeof(bool));
	clock_t start, finish;
	start = clock();
	struct Grids *cuda_grids;

	MyVector *myVector = (struct MyVector *)malloc(sizeof(struct MyVector) * N);
	Points *points = (struct Points *)malloc(sizeof(struct Points) * N);
	Point *Rx = (struct Point *)malloc(sizeof(struct Point));
	Rx->x = RX_X;
	Rx->y = RX_Y;
	Rx->row = int(RX_Y / LENGTH);
	Rx->col = int(RX_X / LENGTH);

	for (int i = 0; i < N; i++) {
		//myVector[i]= (struct MyVector *)malloc(sizeof(struct MyVector));
		myVector[i].x = TX_X;
		myVector[i].y = TX_Y;
		double angle = 2 * PIs / N * i;
		myVector[i].vectorX = cos(angle);
		myVector[i].vectorY = sin(angle);
		myVector[i].col = 0;
		myVector[i].row = 1;
		points[i].N = 0;
	}
	MyVector *cuda_myVector;
	Points *cuda_points;
	Point *cuda_Rx;
	hipMalloc(&cuda_grids, sizeof(struct Grids));
	hipMalloc(&cuda_myVector, sizeof(struct MyVector) * N);
	hipMalloc(&cuda_points, sizeof(struct Points) * N);
	hipMalloc(&cuda_Rx, sizeof(struct Point));

	hipMemcpy(cuda_grids, grids, sizeof(struct Grids), hipMemcpyHostToDevice);
	hipMemcpy(cuda_myVector, myVector, sizeof(struct MyVector) * N, hipMemcpyHostToDevice);
	//hipMemcpy(cuda_points, points, sizeof(struct Points) * 60, hipMemcpyHostToDevice);
	hipMemcpy(cuda_Rx, Rx, sizeof(struct Point), hipMemcpyHostToDevice);
	int threadsPerBlock = 256;
	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
	judgeIsTouched << <blocksPerGrid, threadsPerBlock >> > (cuda_myVector, cuda_grids, cuda_points, cuda_Rx, N);
	hipMemcpy(points, cuda_points, sizeof(struct Points) * N, hipMemcpyDeviceToHost);

	vector<vector<double>> getResult;
	for (int i = 0; i < N; i++) {
		if (points[i].N != 0) {
			vector<double> results;
			for (int j = 0; j < points[i].N; j++) {
				results.push_back(points[i].point[j].x);
				results.push_back(points[i].point[j].y);
				//printf("%d-%d-%d:%f,%f\n", i, j, 2 * i, points[i].point[j].x, points[i].point[j].y);
			}
			getResult.push_back(results);
		}
	}
	finish = clock();
	double duration = (double)(finish - start) / CLOCKS_PER_SEC;
	printf("%f seconds\n", duration);
	//��Ҫ����point
	hipFree(cuda_grids);
	hipFree(cuda_myVector);
	hipFree(cuda_points);
	hipFree(cuda_Rx);
	free(myVector);
	return getResult;
}

//vector<vector<double>> initCUDAInput(vector<vector<vector<double>>> grid_input, double TX_X, double TX_Y, double RX_X, double RX_Y, int N) {
//	//��Ҫ��ʼ������
//	//grids->�洢����
//	//������������Ϊ50*50��С������ɣ�������������С����Щ��������ȫ����Ϊ��
//	//int N = 180;
//	printf("Start...%d\n", sizeof(bool));
//	clock_t start, finish;
//	start = clock();
//	struct Grids *grids = (struct Grids *)malloc(sizeof(struct Grids));
//	grids->width = 2;
//	grids->height = 2;
//	int i = 0;
//	for (; i < grid_input.size(); i++) {
//		if (grid_input[i].size() != 0) {
//			grids->grids[i].N = grid_input[i].size();
//			grids->grids[i].isContains = true;
//			for (int j = 0; j < grid_input[i].size(); j++) {
//				//edge
//				grids->grids[i].edges[j].xstart = grid_input[i][j][0];
//				grids->grids[i].edges[j].ystart = grid_input[i][j][1];
//				grids->grids[i].edges[j].xend = grid_input[i][j][2];
//				grids->grids[i].edges[j].yend = grid_input[i][j][3];
//				grids->grids[i].edges[j].vectorX = grid_input[i][j][4];
//				grids->grids[i].edges[j].vectorY = grid_input[i][j][5];
//			}
//		}
//		else {
//			grids->grids[i].N = 0;
//			grids->grids[i].isContains = false;
//		}
//	}
//	for (; i < ROW*COL; i++) {
//		grids->grids[i].N = 0;
//		grids->grids[i].isContains = false;
//	}
//	struct Grids *cuda_grids;
//
//	MyVector *myVector = (struct MyVector *)malloc(sizeof(struct MyVector) * N);
//	Points *points = (struct Points *)malloc(sizeof(struct Points) * N);
//	Point *Rx = (struct Point *)malloc(sizeof(struct Point));
//	Rx->x = RX_X;
//	Rx->y = RX_Y;
//	Rx->row = int(RX_Y / LENGTH);
//	Rx->col = int(RX_X / LENGTH);
//
//	for (int i = 0; i < N; i++) {
//		//myVector[i]= (struct MyVector *)malloc(sizeof(struct MyVector));
//		myVector[i].x = TX_X;
//		myVector[i].y = TX_Y;
//		double angle = 2 * PIs / N * i;
//		myVector[i].vectorX = cos(angle);
//		myVector[i].vectorY = sin(angle);
//		myVector[i].col = 0;
//		myVector[i].row = 1;
//		points[i].N = 0;
//	}
//	MyVector *cuda_myVector;
//	Points *cuda_points;
//	Point *cuda_Rx;
//	hipMalloc(&cuda_grids, sizeof(struct Grids));
//	hipMalloc(&cuda_myVector, sizeof(struct MyVector) * N);
//	hipMalloc(&cuda_points, sizeof(struct Points) * N);
//	hipMalloc(&cuda_Rx, sizeof(struct Point));
//
//	hipMemcpy(cuda_grids, grids, sizeof(struct Grids), hipMemcpyHostToDevice);
//	hipMemcpy(cuda_myVector, myVector, sizeof(struct MyVector) * N, hipMemcpyHostToDevice);
//	//hipMemcpy(cuda_points, points, sizeof(struct Points) * 60, hipMemcpyHostToDevice);
//	hipMemcpy(cuda_Rx, Rx, sizeof(struct Point), hipMemcpyHostToDevice);
//	int threadsPerBlock = 256;
//	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
//	judgeIsTouched << <blocksPerGrid, threadsPerBlock >> > (cuda_myVector, cuda_grids, cuda_points, cuda_Rx, N);
//	hipMemcpy(points, cuda_points, sizeof(struct Points) * N, hipMemcpyDeviceToHost);
//
//	vector<vector<double>> getResult;
//	for (int i = 0; i < N; i++) {
//		if (points[i].N != 0) {
//			vector<double> results;
//			for (int j = 0; j < points[i].N; j++) {
//				results.push_back(points[i].point[j].x);
//				results.push_back(points[i].point[j].y);
//				//printf("%d-%d-%d:%f,%f\n", i, j, 2 * i, points[i].point[j].x, points[i].point[j].y);
//			}
//			getResult.push_back(results);
//		}
//	}
//	finish = clock();
//	double duration = (double)(finish - start) / CLOCKS_PER_SEC;
//	printf("%f seconds\n", duration);
//	//��Ҫ����point
//	hipFree(cuda_grids);
//	hipFree(cuda_myVector);
//	hipFree(cuda_points);
//	hipFree(cuda_Rx);
//	return getResult;
//}

//int main()
//{
//	//printf("Start...%d\n", sizeof(bool));
//	vector<vector<vector<double>>> grid_input;
//
//	vector<vector<double>> grid0;
//
//	vector<double> edge0;
//	edge0.push_back(1);
//	edge0.push_back(0.5);
//	edge0.push_back(2);
//	edge0.push_back(0.5);
//	edge0.push_back(0);
//	edge0.push_back(1);
//	grid0.push_back(edge0);
//	vector<double> edge1;
//	edge1.push_back(1);
//	edge1.push_back(1.5);
//	edge1.push_back(2);
//	edge1.push_back(1.5);
//	edge1.push_back(0);
//	edge1.push_back(-1);
//	grid0.push_back(edge1);
//
//	vector<double> edge2;
//	edge2.push_back(1);
//	edge2.push_back(0.5);
//	edge2.push_back(1);
//	edge2.push_back(1.5);
//	edge2.push_back(1);
//	edge2.push_back(0);
//	grid0.push_back(edge2);
//	grid_input.push_back(grid0);
//
//	vector<vector<double>> grid1;
//	vector<double> edge3;
//	edge3.push_back(2);
//	edge3.push_back(0.5);
//	edge3.push_back(3);
//	edge3.push_back(0.5);
//	edge3.push_back(0);
//	edge3.push_back(1);
//	grid1.push_back(edge3);
//	vector<double> edge4;
//	edge4.push_back(2);
//	edge4.push_back(1.5);
//	edge4.push_back(3);
//	edge4.push_back(1.5);
//	edge4.push_back(0);
//	edge4.push_back(-1);
//	grid1.push_back(edge4);
//
//	vector<double> edge5;
//	edge5.push_back(3);
//	edge5.push_back(0.5);
//	edge5.push_back(3);
//	edge5.push_back(1.5);
//	edge5.push_back(-1);
//	edge5.push_back(0);
//	grid1.push_back(edge5);
//	grid_input.push_back(grid1);
//
//	vector<vector<double>> grid2;
//	grid_input.push_back(grid2);
//
//	vector<vector<double>> grid3;
//	vector<double> edge6;
//	edge6.push_back(6.5);
//	edge6.push_back(0.5);
//	edge6.push_back(6.5);
//	edge6.push_back(1.5);
//	edge6.push_back(1);
//	edge6.push_back(0);
//	grid3.push_back(edge6);
//	vector<double> edge7;
//	edge7.push_back(7.5);
//	edge7.push_back(0.5);
//	edge7.push_back(6.5);
//	edge7.push_back(0.5);
//	edge7.push_back(0);
//	edge7.push_back(1);
//	grid3.push_back(edge7);
//
//	vector<double> edge8;
//	edge8.push_back(7.5);
//	edge8.push_back(0.5);
//	edge8.push_back(7.5);
//	edge8.push_back(1.5);
//	edge8.push_back(-1);
//	edge8.push_back(0);
//	grid3.push_back(edge8);
//	vector<double> edge9;
//	edge9.push_back(7.5);
//	edge9.push_back(1.5);
//	edge9.push_back(6.5);
//	edge9.push_back(1.5);
//	edge9.push_back(0);
//	edge9.push_back(-1);
//	grid3.push_back(edge9);
//	grid_input.push_back(grid3);
//
//	initCUDAInput(grid_input, 1, 3, 3, 2, 180);
//	//clock_t start, finish;
//	//start = clock();
//	//struct Grids *grids = (struct Grids *)malloc(sizeof(struct Grids));
//
//	////��Ҫ��������װһ��
//	//grids->width = 2;
//	//grids->height = 2;
//
//	//grids->grids[0].N = 3;
//	//grids->grids[0].isContains = true;
//
//	//grids->grids[0].edges[0].xstart = 1;
//	//grids->grids[0].edges[0].ystart = 0.5;
//	//grids->grids[0].edges[0].xend = 2;
//	//grids->grids[0].edges[0].yend = 0.5;
//	//grids->grids[0].edges[0].vectorX = 0;
//	//grids->grids[0].edges[0].vectorY = 1;
//
//	//grids->grids[0].edges[1].xstart = 1;
//	//grids->grids[0].edges[1].ystart = 1.5;
//	//grids->grids[0].edges[1].xend = 2;
//	//grids->grids[0].edges[1].yend = 1.5;
//	//grids->grids[0].edges[1].vectorX = 0;
//	//grids->grids[0].edges[1].vectorY = -1;
//
//	//grids->grids[0].edges[2].xstart = 1;
//	//grids->grids[0].edges[2].ystart = 0.5;
//	//grids->grids[0].edges[2].xend = 1;
//	//grids->grids[0].edges[2].yend = 1.5;
//	//grids->grids[0].edges[2].vectorX = 1;
//	//grids->grids[0].edges[2].vectorY = 0;
//
//	//grids->grids[1].N = 3;
//	//grids->grids[1].isContains = true;
//
//	//grids->grids[1].edges[0].xstart = 2;
//	//grids->grids[1].edges[0].ystart = 0.5;
//	//grids->grids[1].edges[0].xend = 3;
//	//grids->grids[1].edges[0].yend = 0.5;
//	//grids->grids[1].edges[0].vectorX = 0;
//	//grids->grids[1].edges[0].vectorY = 1;
//
//	//grids->grids[1].edges[1].xstart = 2;
//	//grids->grids[1].edges[1].ystart = 1.5;
//	//grids->grids[1].edges[1].xend = 3;
//	//grids->grids[1].edges[1].yend = 1.5;
//	//grids->grids[1].edges[1].vectorX = 0;
//	//grids->grids[1].edges[1].vectorY = -1;
//
//	//grids->grids[1].edges[2].xstart = 3;
//	//grids->grids[1].edges[2].ystart = 0.5;
//	//grids->grids[1].edges[2].xend = 3;
//	//grids->grids[1].edges[2].yend = 1.5;
//	//grids->grids[1].edges[2].vectorX = -1;
//	//grids->grids[1].edges[2].vectorY = 0;
//
//	//grids->grids[2].N = 0;
//	//grids->grids[2].isContains = false;
//
//	//grids->grids[3].N = 4;
//	//grids->grids[3].isContains = true;
//
//	//grids->grids[3].edges[0].xstart = 2.5;
//	//grids->grids[3].edges[0].ystart = 2.5;
//	//grids->grids[3].edges[0].xend = 2.5;
//	//grids->grids[3].edges[0].yend = 3.5;
//	//grids->grids[3].edges[0].vectorX = 1;
//	//grids->grids[3].edges[0].vectorY = 0;
//
//	//grids->grids[3].edges[1].xstart = 3.5;
//	//grids->grids[3].edges[1].ystart = 2.5;
//	//grids->grids[3].edges[1].xend = 3.5;
//	//grids->grids[3].edges[1].yend = 3.5;
//	//grids->grids[3].edges[1].vectorX = -1;
//	//grids->grids[3].edges[1].vectorY = 0;
//
//	//grids->grids[3].edges[2].xstart = 2.5;
//	//grids->grids[3].edges[2].ystart = 2.5;
//	//grids->grids[3].edges[2].xend = 3.5;
//	//grids->grids[3].edges[2].yend = 2.5;
//	//grids->grids[3].edges[2].vectorX = 0;
//	//grids->grids[3].edges[2].vectorY = 1;
//
//	//grids->grids[3].edges[3].xstart = 2.5;
//	//grids->grids[3].edges[3].ystart = 3.5;
//	//grids->grids[3].edges[3].xend = 3.5;
//	//grids->grids[3].edges[3].yend = 3.5;
//	//grids->grids[3].edges[3].vectorX = 0;
//	//grids->grids[3].edges[3].vectorY = -1;
//
//	/*
//	�����ʼ��������ͼ��ʾ��һ��ģ�ͣ����������һ�������
//		 __ __ __ __ __ __ __ __
//		|               __ __   |
//		|     ��       |     |  |
//		|              |__ __|  |
//		|                       |
//		|      __ __ __ __      |
//		|     |           |     |
//		|     |__ __ __ __|     |
//		|__ __ __ __ __ __ __ __|
//	
//	��ֻ�ǲ���һ���ܷ���GPU���㷴�������
//	���������ܹ�ʹ�õģ�����Ϊɶ��ʹ��CPU����û�ж������
//	*/
//	/*grids.grids = new Grid[4];
//	for (int i = 0; i < 4; i++) {
//		grids.grids[i].N = 4;
//		grids.grids[i].isContains = true;
//		grids.grids[i].edges = new Edge[4];
//
//		for (int j = 0; j < 4; j++) {
//			grids.grids[i].edges[j].xstart = 0;
//			grids.grids[i].edges[j].ystart = 0 + i;
//			grids.grids[i].edges[j].xend = 0;
//			grids.grids[i].edges[j].yend = 1 + j;
//			grids.grids[i].edges[j].vectorX = 1;
//			grids.grids[i].edges[j].vectorY = 1;
//		}
//	}*/
//
//	//printf("grids's size%d\n", sizeof(struct Grids));
//	//struct Grids *cuda_grids;
//	//int N = 180;
//	//MyVector *myVector = (struct MyVector *)malloc(sizeof(struct MyVector) * N);
//	//Points *points = (struct Points *)malloc(sizeof(struct Points) * N);
//	//Point *Rx = (struct Point *)malloc(sizeof(struct Point));
//	//Rx->x = 3;
//	//Rx->y = 3;
//
//	//for (int i = 0; i < N; i++) {
//	//	//myVector[i]= (struct MyVector *)malloc(sizeof(struct MyVector));
//	//	myVector[i].x = 1;
//	//	myVector[i].y = 3;
//	//	double angle = 2 * PIs / N * i;
//	//	myVector[i].vectorX = cos(angle);
//	//	myVector[i].vectorY = sin(angle);
//	//	myVector[i].col = 0;
//	//	myVector[i].row = 1;
//	//	points[i].N = 0;
//	//}
//	//MyVector *cuda_myVector;
//	//Points *cuda_points;
//	//Point *cuda_Rx;
//	//hipMalloc(&cuda_grids, sizeof(struct Grids));
//	//hipMalloc(&cuda_myVector, sizeof(struct MyVector) * N);
//	//hipMalloc(&cuda_points, sizeof(struct Points) * N);
//	//hipMalloc(&cuda_Rx, sizeof(struct Point));
//
//	//hipMemcpy(cuda_grids, grids, sizeof(struct Grids), hipMemcpyHostToDevice);
//	//hipMemcpy(cuda_myVector, myVector, sizeof(struct MyVector) * N, hipMemcpyHostToDevice);
//	////hipMemcpy(cuda_points, points, sizeof(struct Points) * 60, hipMemcpyHostToDevice);
//	//hipMemcpy(cuda_Rx, Rx, sizeof(struct Point), hipMemcpyHostToDevice);
//
//
//	//int threadsPerBlock = 256;
//	//int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
//	//judgeIsTouched <<<blocksPerGrid, threadsPerBlock >>> (cuda_myVector, cuda_grids, cuda_points, cuda_Rx, N);
//	//hipMemcpy(points, cuda_points, sizeof(struct Points) * N, hipMemcpyDeviceToHost);
//
//
//	//for (int i = 0; i < N; i++) {
//	//	for (int j = 0; j < points[i].N; j++) {
//	//		printf("%d-%d-%d:%f,%f\n",i,j, 2*i,points[i].point[j].x, points[i].point[j].y);
//	//	}
//	//}
//	//finish = clock();
//	//double duration = (double)(finish - start) / CLOCKS_PER_SEC;
//	//printf("%f seconds\n", duration);
//	//hipFree(cuda_grids);
//	//hipFree(cuda_myVector);
//	//hipFree(cuda_points);
//	//hipFree(cuda_Rx);
//
//
//	//for (int i = 0; i < 4; i++) {
//	//	for (int j = 0; j < 4; j++) {
//	//		printf("%d\n", grids.grids[i].edges[j].xstart);
//	//		//std::cout << grids.grids[i].edges[j].xstart << "," << grids.grids[i].edges[j].ystart << "," << grids.grids[i].edges[j].xend << "," << grids.grids[i].edges[j].yend << std::endl;
//	//	}
//	//}
//	//hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//	//if (cudaStatus != hipSuccess) {
//	//	fprintf(stderr, "addWithCuda failed!");
//	//	return 1;
//	//}
//
//	//printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//	//	c[0], c[1], c[2], c[3], c[4]);
//
//	//// hipDeviceReset must be called before exiting in order for profiling and
//	//// tracing tools such as Nsight and Visual Profiler to show complete traces.
//	//cudaStatus = hipDeviceReset();
//	//if (cudaStatus != hipSuccess) {
//	//	fprintf(stderr, "hipDeviceReset failed!");
//	//	return 1;
//	//}
//
//	return 0;
//}
//
