#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "time.h" 
#include "MyCUDARayTracing.h"
#include "StaticConstants.h"
#include <QtCore/QDebug>



//������ʹ�õ�λ����
/*
���ǻ���һ���ļ��裬��Ϊ������ܳ��Ⱥ��ܿ�����
ÿ��������Ŀ���趨Ϊ2
�������֪����ǰ����Ϊ(x,y)
(int(x/2),int(y/2))��ʾ��ǰ���ڵ�����
��һά����grid������Ϊ int(x/2)+int(y/2)*ROW
����������Ҫ����������Ϣ����ָ����ǰ����λ�õĴ��䷽��
	������ڸ��ӱ߽紦������Ҫ�ı�
	����ں��ݽ��㴦
		��һ����	����
		�ڶ�����	(int(x/2)-1)+int(y/2)*ROW
		��������	(int(x/2)-1)+(int(y/2)-1)*ROW
		��������	(int(x/2))-1+int(y/2)*ROW
	����ں���߽紦
		ָ���������� int(x/2)+(int(y/2)-1)*ROW
	���������߽紦
		ָ��������� (int(x/2))-1+int(y/2)*ROW
*/



__device__ double directionIntersect(double x1, double y1, double x2, double y2, double x3, double y3) {
	//(x3-x1,y3-y1)X(x2-x1,y2-y1)
	return (y2 - y1)*(x3 - x1) - (x2 - x1)*(y3 - y1);
}
__device__ double Mymin(double x1, double x2) {
	if (x1 < x2)
		return x1;
	else
		return x2;
}
__device__ double Mymax(double x1, double x2) {
	return x1 > x2 ? x1 : x2;
}
__device__ double calDistance(double x1, double y1, double x2, double y2) {
	return sqrt((x2 - x1)*(x2 - x1) + (y2 - y1)*(y2 - y1));
}
__device__ bool onSegment(double x1, double y1, double x2, double y2, double x3, double y3) {
	if (Mymin(x1, x2) <= x3 && x3 < Mymax(x1, x2) && Mymin(y1, y2) < y3&&y3 < Mymax(y1, y2)) {
		return true;
	}
	else {
		return false;
	}
}
__device__ void calculPointOfIntersection(double x1, double y1, double x2, double y2, double x3, double y3, double x4, double y4, double &x, double &y) {
	double a = y2 - y1;
	double b = x1 - x2;
	double c = y4 - y3;
	double d = x3 - x4;
	double e = (y2 - y1)*x1 - (x2 - x1)*y1;
	double f = (y4 - y3)*x3 - (x4 - x3)*y3;
	x = (e*d - b * f) / (a*d - b * c);
	y = (a*f - e * c) / (a*d - b * c);
}
__device__ bool segmentsIntersect(double x1, double y1, double x2, double y2, double x3, double y3, double x4, double y4) {
	double d1 = directionIntersect(x3, y3, x4, y4, x1, y1);
	double d2 = directionIntersect(x3, y3, x4, y4, x2, y2);
	double d3 = directionIntersect(x1, y1, x2, y2, x3, y3);
	double d4 = directionIntersect(x1, y1, x2, y2, x4, y4);
	if (((d1 > 0 && d2 < 0) || (d1 < 0 && d2>0)) && ((d3 > 0 && d4 < 0) || (d3 < 0 && d4>0)))
		return true;
	else if (d1 == 0 && onSegment(x3, y3, x4, y4, x1, y1)) {
		return true;
	}
	else if (d2 == 0 && onSegment(x3, y3, x4, y4, x2, y2)) {
		return true;
	}
	else if (d3 == 0 && onSegment(x1, y1, x2, y2, x3, y3)) {
		return true;
	}
	else if (d4 == 0 && onSegment(x1, y1, x2, y2, x4, y4)) {
		return true;
	}
	else
		return false;
}

__device__ void normalizeVector(double &x, double &y) {
	double norm = sqrt(x*x + y * y);
	x = x / norm;
	y = y / norm;
}
//[x1,y1],[x2,y2]���ɵ�ֱ�ߵ�[x0,y0]�ľ���
__device__ double calculDistancePointAndLine(double x1, double y1, double x2, double y2, double x0, double y0) {
	double A = y2 - y1;
	double B = x1 - x2;
	double C = x2 * y1 - x1 * y2;
	return abs(A*x0 + B * y0 + C) / sqrt(A*A + B * B);
}

__device__ bool isEndPoint(Edge edge, double x, double y) {
	if (calDistance(edge.xend, edge.yend, x, y) < IS_DIFFRACTION_RADIUS || calDistance(edge.xstart, edge.ystart, x, y) < IS_DIFFRACTION_RADIUS) {
		return true;
	}
	else {
		return false;
	}
}

//__global__ void diffractionJudgeIsTouched(MyVector *vector, Grids *grids, Points *points, Point *point,int N) {
//	int i = blockDim.x * blockIdx.x + threadIdx.x;
//	if (i < N) {
//		int row = vector[i].row;
//		int col = vector[i].col;
//		int oldRow = row;
//		int oldCol = col;
//		double x = vector[i].x;
//		double y = vector[i].y;
//		double vectorX = vector[i].vectorX;
//		double vectorY = vector[i].vectorY;
//		while (true) {
//			oldRow = row;
//			oldCol = col;
//			if (row < 0 || col < 0 || row >= ROW || col >= COL)
//				break;
//			int indexC = row * COL + col;
//			//���ȼ���������һ���ߵĽ��㹹��һ���߶�
//			int currentX = col * LENGTH;
//			int currentY = row * LENGTH;
//			//��Ҫ�ж��Ƿ����
//			double newX = x, newY = y;
//			if (vectorX > 0 && vectorY > 0) {
//				if ((currentX + LENGTH - x)*vectorY > (currentY + LENGTH - y)*vectorX) {
//					//top
//					newX = x + (currentY + LENGTH - y)*vectorX / vectorY;
//					newY = currentY + LENGTH;
//					row++;
//				}
//				else if ((currentX + LENGTH - x)*vectorY == (currentY + LENGTH - y)*vectorX) {
//					newX = currentX + LENGTH;
//					newY = currentY + LENGTH;
//					row++;
//					col++;
//				}
//				else {
//					//right
//					newX = currentX + LENGTH;
//					newY = vectorY / vectorX * (currentX + 2 - x) + y;
//					col++;
//				}
//			}
//			else if (vectorX > 0 && vectorY < 0) {
//				if (-vectorY * (currentX + LENGTH - x) < vectorX*(y - currentY)) {
//					//right
//					newY = y + (currentX + 2 - x)*vectorY / vectorX;
//					newX = currentX + LENGTH;
//					col++;
//				}
//				else if (-vectorY * (currentX + LENGTH - x) == vectorX * (y - currentY)) {
//					newX = currentX + LENGTH;
//					newY = currentY;
//					col++;
//					row--;
//				}
//				else {
//					//bottom
//					newX = x - vectorX / vectorY * (y - currentY);
//					newY = currentY;
//					row--;
//				}
//			}
//			else if (vectorX < 0 && vectorY>0) {
//				if (-vectorX * (currentY + LENGTH - y) < vectorY*(x - currentX)) {
//					//top
//					newX = x + (vectorX / vectorY * (currentY + LENGTH - y));
//					newY = currentY + LENGTH;
//					row++;
//				}
//				else if (-vectorX * (currentY + LENGTH - y) == vectorY * (x - currentX)) {
//					newX = currentX;
//					newY = currentY + LENGTH;
//					row++;
//					col--;
//				}
//				else {
//					//left
//					newY = y + (vectorY / vectorX * (currentX - x));
//					newX = currentX;
//					col--;
//				}
//			}
//			else if (vectorX < 0 && vectorY < 0) {
//				if (vectorY*(x - currentX) < vectorX*(y - currentY)) {
//					//bottom
//					newX = x - vectorX / vectorY * (y - currentY);
//					newY = currentY;
//					row--;
//				}
//				else if (vectorY*(x - currentX) == vectorX * (y - currentY)) {
//					newX = currentX;
//					newY = currentY;
//					row--;
//					col--;
//				}
//				else {
//					//left
//					newY = y - (x - currentX)*vectorY / vectorX;
//					newX = currentX;
//					col--;
//				}
//			}
//			else if (vectorX == 0) {
//				newY = vectorY > 0 ? currentY + LENGTH : currentY;
//				row = vectorY > 0 ? row + 1 : row - 1;
//			}
//			else if (vectorY == 0) {
//				newX = vectorX > 0 ? currentX + LENGTH : currentY;
//				col = vectorX > 0 ? col + 1 : col - 1;
//			}
//			//vectorX vectorY������ͬʱΪ0
//			//��������а����߻��߽��յ�λ�ڸø�����ʱ������Ҫ����
//			if (grids->grids[indexC].isContains || (oldRow == point->row&&oldCol == point->col)) {
//				//contains edge
//				double distanceM = std::sqrt(2.0)*LENGTH + 1;
//				bool isFind = false;
//				double insertPointOutX = 0, insertPointOutY = 0;
//				int edgeIndex = -1;
//
//				for (int k = 0; k < grids->grids[indexC].N; k++) {
//					if (segmentsIntersect(x, y, newX, newY, grids->grids[indexC].edges[k].xstart, grids->grids[indexC].edges[k].ystart, grids->grids[indexC].edges[k].xend, grids->grids[indexC].edges[k].yend) == true) {
//						double insertPointX = 0, insertPointY = 0;//��ǰ����
//						calculPointOfIntersection(x, y, newX, newY, grids->grids[indexC].edges[k].xstart, grids->grids[indexC].edges[k].ystart, grids->grids[indexC].edges[k].xend, grids->grids[indexC].edges[k].yend, insertPointX, insertPointY);
//						if (currentX > insertPointX || insertPointX > currentX + LENGTH || currentY > insertPointY || insertPointY > currentY + LENGTH) {
//							//������������������
//							continue;
//						}
//						double tempDistance = calDistance(x, y, insertPointX, insertPointY);
//						//��ô���õ�Ŀ���Ǵ��ڷ�����µ������Ȼ����������ڣ�������һ��ǽ���ϣ��ű�Ȼ�����ǽ���ཻ�ģ������Ҫ���˵��������
//						//��ֹ�������Լ�
//						if (abs(insertPointX - x) < 0.0001 && abs(insertPointY - y) < 0.0001)
//							continue;
//						if (tempDistance < distanceM) {
//							isFind = true;
//							distanceM = tempDistance;
//							insertPointOutX = insertPointX;
//							insertPointOutY = insertPointY;
//							edgeIndex = k;
//						}
//					}
//				}
//				bool isReached = false;
//				if (oldRow == point->row&&oldCol == point->col) {
//					//��Ŀ��������������ʱ����Ҫ�ж��Ƿ���Ŀ����ཻ
//					if (x != newX && y != newY) {
//						double targetDistance = calculDistancePointAndLine(x, y, newX, newY, point->x, point->y);
//						//ֻ���ڲ���Բ�뾶���Ҿ����������������н���
//						if (targetDistance < CAPTURE_RADIUS) {
//							double targetPointDistance = calDistance(x, y, point->x, point->y);
//							if (targetPointDistance < distanceM) {
//								distanceM = targetPointDistance;
//								insertPointOutX = point->x;
//								insertPointOutY = point->y;
//								isReached = true;
//								isFind = true;
//							}
//						}
//					}
//				}
//
//				if (isFind&&isReached) {
//					//��Ŀ����ཻ����������
//					points[i].point[points[i].N].x = insertPointOutX;
//					points[i].point[points[i].N].y = insertPointOutY;
//					points[i].N++;
//					points[i].isFind = true;
//					break;
//				}
//				//�Ѿ���������ˣ�����н��㣬����Ҫ�������ߴ���ķ���
//				//����������䣬��Ҫ�ڵ�ǰ������һ���µķ������
//				else if (isFind) {
//					//��ǰ���ߵ�����
//					double currentVectorX = x - insertPointOutX;
//					double currentVectorY = y - insertPointOutY;
//					//Ҫ��������
//					//���������ǶԳ�
//					// \  |  /
//					//  \ | /
//					//   \|/
//					//------------------
//					//����ǽ��ķ�����
//					//grids->grids[indexC].edges[k]
//					//֮ǰ���ǽ��ķ�������˳ʱ�뷽��ģ�Ҳ���ǳ��ڣ���������Ĭ������ֻ���ⲿ�����
//					//��˷�������ȡ��
//					double edgeVectorX = -grids->grids[indexC].edges[edgeIndex].vectorX;
//					double edgeVectorY = -grids->grids[indexC].edges[edgeIndex].vectorY;
//
//					if (edgeVectorX*currentVectorX + edgeVectorY * currentVectorY < 0) {
//						//�н��������90������Ҫת��
//						edgeVectorX = -edgeVectorX;
//						edgeVectorY = -edgeVectorY;
//					}
//					//ע�⣬ǽ��ķ�����Ϊ��λ����
//					//���������ڷ������ϵ�ͶӰΪ(uv)v 
//					//��������Ϊ2(uv)v-u
//					//�ο�https://www.cnblogs.com/graphics/archive/2013/02/21/2920627.html
//
//					double temp = currentVectorX * edgeVectorX + currentVectorY * edgeVectorY;//uv
//					double edgeProjectVectorX = temp * edgeVectorX;
//					double edgeProjectVectorY = temp * edgeVectorY;
//					//������ϣ��������ݣ�������д�����Points
//					vectorX = 2 * edgeProjectVectorX - currentVectorX;
//					vectorY = 2 * edgeProjectVectorY - currentVectorY;
//					normalizeVector(vectorX, vectorY);//�����Ժ���Ҫ��һ��
//					x = insertPointOutX;//���·������
//					y = insertPointOutY;
//					points[i].point[points[i].N].x = x;
//					points[i].point[points[i].N].y = y;
//					points[i].N++;
//					row = oldRow;
//					col = oldCol;
//					if (points[i].N > MAX_REFLECTION_TIMES) {
//						//������������,û�ҵ����㣬��˵���Ϊ0
//						points[i].N = 0;
//						points[i].isFind = false;
//						break;
//					}
//					//}
//
//				}
//				else {
//					x = newX;
//					y = newY;
//				}
//			}
//			else {
//				//��������
//				x = newX;
//				y = newY;
//			}
//			//printf("current position:%f,%f", x, y);
//			//�����newX��newY�ǽ�����������꣬��ʱ��Ҫ�жϸ����߶�������еı��Ƿ��ཻ
//			//break;
//			//if (grids[indexC].grids->isContains) {
//			//	
//			//	for (int k = 0; k < grids[indexC].grids->N; i++) {
//			//		//judge weather is intersecting
//			//	}
//
//			//}
//			//else {
//
//
//			//	//��Ҫ�ҵ��������ĸ���
//			//	//update vectors
//			//	//remain direction update x and y
//			//}
//		}
//		//vector[i].
//	}
//}
__global__ void judegTouchedDiffraction(MyVector *vector, Grids *grids, Points *diffractionPoints, Point *point, int N, int currentPointsIndex) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < N) {
		int row = vector[i].row;
		int col = vector[i].col;
		int oldRow = row;
		int oldCol = col;
		double x = vector[i].x;
		double y = vector[i].y;
		double vectorX = vector[i].vectorX;
		double vectorY = vector[i].vectorY;
		while (true) {
			oldRow = row;
			oldCol = col;
			if (row < 0 || col < 0 || row >= ROW || col >= COL)
				break;
			int indexC = row * COL + col;
			//���ȼ���������һ���ߵĽ��㹹��һ���߶�
			int currentX = col * LENGTH;
			int currentY = row * LENGTH;
			//��Ҫ�ж��Ƿ����
			double newX = x, newY = y;
			if (vectorX > 0 && vectorY > 0) {
				if ((currentX + LENGTH - x)*vectorY > (currentY + LENGTH - y)*vectorX) {
					//top
					newX = x + (currentY + LENGTH - y)*vectorX / vectorY;
					newY = currentY + LENGTH;
					row++;
				}
				else if ((currentX + LENGTH - x)*vectorY == (currentY + LENGTH - y)*vectorX) {
					newX = currentX + LENGTH;
					newY = currentY + LENGTH;
					row++;
					col++;
				}
				else {
					//right
					newX = currentX + LENGTH;
					newY = vectorY / vectorX * (currentX + 2 - x) + y;
					col++;
				}
			}
			else if (vectorX > 0 && vectorY < 0) {
				if (-vectorY * (currentX + LENGTH - x) < vectorX*(y - currentY)) {
					//right
					newY = y + (currentX + 2 - x)*vectorY / vectorX;
					newX = currentX + LENGTH;
					col++;
				}
				else if (-vectorY * (currentX + LENGTH - x) == vectorX * (y - currentY)) {
					newX = currentX + LENGTH;
					newY = currentY;
					col++;
					row--;
				}
				else {
					//bottom
					newX = x - vectorX / vectorY * (y - currentY);
					newY = currentY;
					row--;
				}
			}
			else if (vectorX < 0 && vectorY>0) {
				if (-vectorX * (currentY + LENGTH - y) < vectorY*(x - currentX)) {
					//top
					newX = x + (vectorX / vectorY * (currentY + LENGTH - y));
					newY = currentY + LENGTH;
					row++;
				}
				else if (-vectorX * (currentY + LENGTH - y) == vectorY * (x - currentX)) {
					newX = currentX;
					newY = currentY + LENGTH;
					row++;
					col--;
				}
				else {
					//left
					newY = y + (vectorY / vectorX * (currentX - x));
					newX = currentX;
					col--;
				}
			}
			else if (vectorX < 0 && vectorY < 0) {
				if (vectorY*(x - currentX) < vectorX*(y - currentY)) {
					//bottom
					newX = x - vectorX / vectorY * (y - currentY);
					newY = currentY;
					row--;
				}
				else if (vectorY*(x - currentX) == vectorX * (y - currentY)) {
					newX = currentX;
					newY = currentY;
					row--;
					col--;
				}
				else {
					//left
					newY = y - (x - currentX)*vectorY / vectorX;
					newX = currentX;
					col--;
				}
			}
			else if (vectorX == 0) {
				newY = vectorY > 0 ? currentY + LENGTH : currentY;
				row = vectorY > 0 ? row + 1 : row - 1;
			}
			else if (vectorY == 0) {
				newX = vectorX > 0 ? currentX + LENGTH : currentY;
				col = vectorX > 0 ? col + 1 : col - 1;
			}
			//vectorX vectorY������ͬʱΪ0
			//��������а����߻��߽��յ�λ�ڸø�����ʱ������Ҫ����
			if (grids->grids[indexC].isContains || (oldRow == point->row&&oldCol == point->col)) {
				//contains edge
				double distanceM = std::sqrt(2.0)*LENGTH + 1;
				bool isFind = false;
				double insertPointOutX = 0, insertPointOutY = 0;
				int edgeIndex = -1;

				for (int k = 0; k < grids->grids[indexC].N; k++) {
					if (segmentsIntersect(x, y, newX, newY, grids->grids[indexC].edges[k].xstart, grids->grids[indexC].edges[k].ystart, grids->grids[indexC].edges[k].xend, grids->grids[indexC].edges[k].yend) == true) {
						double insertPointX = 0, insertPointY = 0;//��ǰ����
						calculPointOfIntersection(x, y, newX, newY, grids->grids[indexC].edges[k].xstart, grids->grids[indexC].edges[k].ystart, grids->grids[indexC].edges[k].xend, grids->grids[indexC].edges[k].yend, insertPointX, insertPointY);
						if (currentX > insertPointX || insertPointX > currentX + LENGTH || currentY > insertPointY || insertPointY > currentY + LENGTH) {
							//������������������
							continue;
						}
						double tempDistance = calDistance(x, y, insertPointX, insertPointY);
						//��ô���õ�Ŀ���Ǵ��ڷ�����µ������Ȼ����������ڣ�������һ��ǽ���ϣ��ű�Ȼ�����ǽ���ཻ�ģ������Ҫ���˵��������
						//��ֹ�������Լ�
						if (abs(insertPointX - x) < 0.0001 && abs(insertPointY - y) < 0.0001)
							continue;
						if (tempDistance < distanceM) {
							isFind = true;
							distanceM = tempDistance;
							insertPointOutX = insertPointX;
							insertPointOutY = insertPointY;
							edgeIndex = k;
						}
					}
				}
				bool isReached = false;
				if (oldRow == point->row&&oldCol == point->col) {
					//��Ŀ��������������ʱ����Ҫ�ж��Ƿ���Ŀ����ཻ
					if (x != newX && y != newY) {
						double targetDistance = calculDistancePointAndLine(x, y, newX, newY, point->x, point->y);
						//ֻ���ڲ���Բ�뾶���Ҿ����������������н���
						if (targetDistance < CAPTURE_RADIUS) {
							double targetPointDistance = calDistance(x, y, point->x, point->y);
							if (targetPointDistance < distanceM) {
								distanceM = targetPointDistance;
								insertPointOutX = point->x;
								insertPointOutY = point->y;
								isReached = true;
								isFind = true;
							}
						}
					}
				}

				if (isFind&&isReached) {
					//��Ŀ����ཻ����������
					diffractionPoints[currentPointsIndex*N + i].point[diffractionPoints[currentPointsIndex*N + i].N].x = insertPointOutX;
					diffractionPoints[currentPointsIndex*N + i].point[diffractionPoints[currentPointsIndex*N + i].N].y = insertPointOutY;
					diffractionPoints[currentPointsIndex*N + i].N++;
					diffractionPoints[currentPointsIndex*N + i].isFind = true;
					break;
				}
				//�Ѿ���������ˣ�����н��㣬����Ҫ�������ߴ���ķ���
				//����������䣬��Ҫ�ڵ�ǰ������һ���µķ������
				else if (isFind) {
					//�ڴ˴��ж��Ƿ�������
					//isDiffraction����Ϊfalse���ܷ��ʸô�
					//if ((!isDiffraction) && isEndPoint(grids->grids[indexC].edges[edgeIndex], insertPointOutX, insertPointOutY)) {
					//	if (calDistance(grids->grids[indexC].edges[edgeIndex].xend, grids->grids[indexC].edges[edgeIndex].yend, insertPointOutX, insertPointOutY) < calDistance(grids->grids[indexC].edges[edgeIndex].xstart, grids->grids[indexC].edges[edgeIndex].ystart, insertPointOutX, insertPointOutY)) {
					//		insertPointOutX = grids->grids[indexC].edges[edgeIndex].xend;
					//		insertPointOutY = grids->grids[indexC].edges[edgeIndex].yend;
					//	}
					//	else {
					//		insertPointOutX = grids->grids[indexC].edges[edgeIndex].xstart;
					//		insertPointOutY = grids->grids[indexC].edges[edgeIndex].ystart;
					//	}
					//	//���������䣬�����µ��߳�
					//	MyVector *myVectorNew = (struct MyVector *)malloc(sizeof(struct MyVector) * N);
					//	points[i].point[points[i].N].x = insertPointOutX;
					//	points[i].point[points[i].N].y = insertPointOutY;
					//	points[i].N++;
					//	if (points[i].N > MAX_REFLECTION_TIMES) {
					//		//������������,û�ҵ����㣬��˵���Ϊ0
					//		points[i].N = 0;
					//		points[i].isFind = false;
					//		break;
					//	}
					//	else {
					//		points[i].isFind = true;
					//	}
					//	for (int m = 0; m < N; m++) {
					//		//myVector[i]= (struct MyVector *)malloc(sizeof(struct MyVector));
					//		//��Ҫ�������д��points

					//		myVectorNew[m].x = insertPointOutX;
					//		myVectorNew[m].y = insertPointOutY;
					//		double angle = 2 * PIs / N * i;
					//		myVectorNew[m].vectorX = cos(angle);
					//		myVectorNew[m].vectorY = sin(angle);
					//		myVectorNew[m].col = int(insertPointOutX / 2);
					//		myVectorNew[m].row = int(insertPointOutY / 2);
					//		//MyVector *cuda_myVectorNew;
					//		//hipMalloc(&cuda_myVectorNew, sizeof(struct MyVector) * N);
					//		//hipMemcpy(cuda_myVectorNew, myVectorNew, sizeof(struct MyVector) * N, hipMemcpyHostToDevice);


					//	}
					//	int threadsPerBlock = 90;
					//	int blocksPerGrid = N / 90;
					//	//judgeIsTouched << <blocksPerGrid, threadsPerBlock >> > (myVectorNew, grids, points, diffractionPoints, point, N, true, i);
					//	//ִ�н����Ժ󽫲�������ִ��
					//	break;
					//}
					//else {
						//Severity	Code	Description	Project	File	Line	Suppression State
						//Error		kernel launch from __device__ or __global__ functions requires separate compilation mode	Demo1	D : \program\qt\R\RayTracingCUDA\raytracing3.0\Demo1\MyRayTracingCUDA.cu	574

						//��ǰ���ߵ�����
					double currentVectorX = x - insertPointOutX;
					double currentVectorY = y - insertPointOutY;
					//Ҫ��������
					//���������ǶԳ�
					// \  |  /
					//  \ | /
					//   \|/
					//------------------
					//����ǽ��ķ�����
					//grids->grids[indexC].edges[k]
					//֮ǰ���ǽ��ķ�������˳ʱ�뷽��ģ�Ҳ���ǳ��ڣ���������Ĭ������ֻ���ⲿ�����
					//��˷�������ȡ��
					double edgeVectorX = grids->grids[indexC].edges[edgeIndex].vectorX;
					double edgeVectorY = grids->grids[indexC].edges[edgeIndex].vectorY;

					if (edgeVectorX*currentVectorX + edgeVectorY * currentVectorY < 0) {
						//˵��������ǽ���ڲ������ֱ�ӽ���
						//��������
						//�н��������90������Ҫת��
						//edgeVectorX = -edgeVectorX;
						//edgeVectorY = -edgeVectorY;
						diffractionPoints[currentPointsIndex*N + i].N = 0;
						diffractionPoints[currentPointsIndex*N + i].isFind = false;
						break;
					}
					else {
						double temp = currentVectorX * edgeVectorX + currentVectorY * edgeVectorY;//uv
						double edgeProjectVectorX = temp * edgeVectorX;
						double edgeProjectVectorY = temp * edgeVectorY;
						//������ϣ��������ݣ�������д�����Points
						vectorX = 2 * edgeProjectVectorX - currentVectorX;
						vectorY = 2 * edgeProjectVectorY - currentVectorY;
						normalizeVector(vectorX, vectorY);//�����Ժ���Ҫ��һ��
						x = insertPointOutX;//���·������
						y = insertPointOutY;
						//if (isDiffraction) {
							//˵���������������,��Ҫ������д����Ӧ��λ��
						diffractionPoints[currentPointsIndex*N + i].point[diffractionPoints[currentPointsIndex*N + i].N].x = x;
						diffractionPoints[currentPointsIndex*N + i].point[diffractionPoints[currentPointsIndex*N + i].N].y = y;
						diffractionPoints[currentPointsIndex*N + i].N++;
						if (diffractionPoints[currentPointsIndex*N + i].N > MAX_REFLECTION_TIMES_AFTER_DIFFRACTION) {
							diffractionPoints[currentPointsIndex*N + i].N = 0;
							diffractionPoints[currentPointsIndex*N + i].isFind = false;
							break;
						}
						//}
						//else {
						//	points[i].point[points[i].N].x = x;
						//	points[i].point[points[i].N].y = y;
						//	points[i].N++;
						//	if (points[i].N > MAX_REFLECTION_TIMES) {
						//		//������������,û�ҵ����㣬��˵���Ϊ0
						//		points[i].N = 0;
						//		points[i].isFind = false;
						//		break;
						//	}
						//}
						row = oldRow;
						col = oldCol;
					}
					//ע�⣬ǽ��ķ�����Ϊ��λ����
					//���������ڷ������ϵ�ͶӰΪ(uv)v 
					//��������Ϊ2(uv)v-u
					//�ο�https://www.cnblogs.com/graphics/archive/2013/02/21/2920627.html


				//}
				//}

				}
				else {
					x = newX;
					y = newY;
				}
			}
			else {
				//��������
				x = newX;
				y = newY;
			}
			//printf("current position:%f,%f", x, y);
			//�����newX��newY�ǽ�����������꣬��ʱ��Ҫ�жϸ����߶�������еı��Ƿ��ཻ
			//break;
			//if (grids[indexC].grids->isContains) {
			//	
			//	for (int k = 0; k < grids[indexC].grids->N; i++) {
			//		//judge weather is intersecting
			//	}

			//}
			//else {


			//	//��Ҫ�ҵ��������ĸ���
			//	//update vectors
			//	//remain direction update x and y
			//}
		}
		//vector[i].
	}
}


/*
vector is input
points is output
points�Ǵ洢������ߵı���
point�Ǵ洢RX��ı���
N�����߷ֱ����
*/
__global__ void judgeIsTouched(MyVector *vector, Grids *grids, Points *points, Points *diffractionPoints, Point *point, int N, int currentPointsIndex) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < N) {
		int row = vector[i].row;
		int col = vector[i].col;
		int oldRow = row;
		int oldCol = col;
		double x = vector[i].x;
		double y = vector[i].y;
		double vectorX = vector[i].vectorX;
		double vectorY = vector[i].vectorY;
		while (true) {
			oldRow = row;
			oldCol = col;
			if (row < 0 || col < 0 || row >= ROW || col >= COL)
				break;
			int indexC = row * COL + col;
			//���ȼ���������һ���ߵĽ��㹹��һ���߶�
			int currentX = col * LENGTH;
			int currentY = row * LENGTH;
			//��Ҫ�ж��Ƿ����
			double newX = x, newY = y;
			if (vectorX > 0 && vectorY > 0) {
				if ((currentX + LENGTH - x)*vectorY > (currentY + LENGTH - y)*vectorX) {
					//top
					newX = x + (currentY + LENGTH - y)*vectorX / vectorY;
					newY = currentY + LENGTH;
					row++;
				}
				else if ((currentX + LENGTH - x)*vectorY == (currentY + LENGTH - y)*vectorX) {
					newX = currentX + LENGTH;
					newY = currentY + LENGTH;
					row++;
					col++;
				}
				else {
					//right
					newX = currentX + LENGTH;
					newY = vectorY / vectorX * (currentX + 2 - x) + y;
					col++;
				}
			}
			else if (vectorX > 0 && vectorY < 0) {
				if (-vectorY * (currentX + LENGTH - x) < vectorX*(y - currentY)) {
					//right
					newY = y + (currentX + 2 - x)*vectorY / vectorX;
					newX = currentX + LENGTH;
					col++;
				}
				else if (-vectorY * (currentX + LENGTH - x) == vectorX * (y - currentY)) {
					newX = currentX + LENGTH;
					newY = currentY;
					col++;
					row--;
				}
				else {
					//bottom
					newX = x - vectorX / vectorY * (y - currentY);
					newY = currentY;
					row--;
				}
			}
			else if (vectorX < 0 && vectorY>0) {
				if (-vectorX * (currentY + LENGTH - y) < vectorY*(x - currentX)) {
					//top
					newX = x + (vectorX / vectorY * (currentY + LENGTH - y));
					newY = currentY + LENGTH;
					row++;
				}
				else if (-vectorX * (currentY + LENGTH - y) == vectorY * (x - currentX)) {
					newX = currentX;
					newY = currentY + LENGTH;
					row++;
					col--;
				}
				else {
					//left
					newY = y + (vectorY / vectorX * (currentX - x));
					newX = currentX;
					col--;
				}
			}
			else if (vectorX < 0 && vectorY < 0) {
				if (vectorY*(x - currentX) < vectorX*(y - currentY)) {
					//bottom
					newX = x - vectorX / vectorY * (y - currentY);
					newY = currentY;
					row--;
				}
				else if (vectorY*(x - currentX) == vectorX * (y - currentY)) {
					newX = currentX;
					newY = currentY;
					row--;
					col--;
				}
				else {
					//left
					newY = y - (x - currentX)*vectorY / vectorX;
					newX = currentX;
					col--;
				}
			}
			else if (vectorX == 0) {
				newY = vectorY > 0 ? currentY + LENGTH : currentY;
				row = vectorY > 0 ? row + 1 : row - 1;
			}
			else if (vectorY == 0) {
				newX = vectorX > 0 ? currentX + LENGTH : currentY;
				col = vectorX > 0 ? col + 1 : col - 1;
			}
			//vectorX vectorY������ͬʱΪ0
			//��������а����߻��߽��յ�λ�ڸø�����ʱ������Ҫ����
			if (grids->grids[indexC].isContains || (oldRow == point->row&&oldCol == point->col)) {
				//contains edge
				double distanceM = std::sqrt(2.0)*LENGTH + 1;
				bool isFind = false;
				double insertPointOutX = 0, insertPointOutY = 0;
				int edgeIndex = -1;

				for (int k = 0; k < grids->grids[indexC].N; k++) {
					if (segmentsIntersect(x, y, newX, newY, grids->grids[indexC].edges[k].xstart, grids->grids[indexC].edges[k].ystart, grids->grids[indexC].edges[k].xend, grids->grids[indexC].edges[k].yend) == true) {
						double insertPointX = 0, insertPointY = 0;//��ǰ����
						calculPointOfIntersection(x, y, newX, newY, grids->grids[indexC].edges[k].xstart, grids->grids[indexC].edges[k].ystart, grids->grids[indexC].edges[k].xend, grids->grids[indexC].edges[k].yend, insertPointX, insertPointY);
						if (currentX > insertPointX || insertPointX > currentX + LENGTH || currentY > insertPointY || insertPointY > currentY + LENGTH) {
							//������������������
							continue;
						}
						double tempDistance = calDistance(x, y, insertPointX, insertPointY);
						//��ô���õ�Ŀ���Ǵ��ڷ�����µ������Ȼ����������ڣ�������һ��ǽ���ϣ��ű�Ȼ�����ǽ���ཻ�ģ������Ҫ���˵��������
						//��ֹ�������Լ�
						if (abs(insertPointX - x) < 0.0001 && abs(insertPointY - y) < 0.0001)
							continue;
						if (tempDistance < distanceM) {
							isFind = true;
							distanceM = tempDistance;
							insertPointOutX = insertPointX;
							insertPointOutY = insertPointY;
							edgeIndex = k;
						}
					}
				}
				bool isReached = false;
				if (oldRow == point->row&&oldCol == point->col) {
					//��Ŀ��������������ʱ����Ҫ�ж��Ƿ���Ŀ����ཻ
					if (x != newX && y != newY) {
						double targetDistance = calculDistancePointAndLine(x, y, newX, newY, point->x, point->y);
						//ֻ���ڲ���Բ�뾶���Ҿ����������������н���
						if (targetDistance < CAPTURE_RADIUS) {
							double targetPointDistance = calDistance(x, y, point->x, point->y);
							if (targetPointDistance < distanceM) {
								distanceM = targetPointDistance;
								insertPointOutX = point->x;
								insertPointOutY = point->y;
								isReached = true;
								isFind = true;
							}
						}
					}
				}

				if (isFind&&isReached) {
					//��Ŀ����ཻ����������
					points[i].point[points[i].N].x = insertPointOutX;
					points[i].point[points[i].N].y = insertPointOutY;
					points[i].N++;
					points[i].isFind = true;
					break;
				}
				//�Ѿ���������ˣ�����н��㣬����Ҫ�������ߴ���ķ���
				//����������䣬��Ҫ�ڵ�ǰ������һ���µķ������
				else if (isFind) {
					//�ڴ˴��ж��Ƿ�������
					//isDiffraction����Ϊfalse���ܷ��ʸô�
					if (isEndPoint(grids->grids[indexC].edges[edgeIndex], insertPointOutX, insertPointOutY)) {
						if (calDistance(grids->grids[indexC].edges[edgeIndex].xend, grids->grids[indexC].edges[edgeIndex].yend, insertPointOutX, insertPointOutY) < calDistance(grids->grids[indexC].edges[edgeIndex].xstart, grids->grids[indexC].edges[edgeIndex].ystart, insertPointOutX, insertPointOutY)) {
							insertPointOutX = grids->grids[indexC].edges[edgeIndex].xend;
							insertPointOutY = grids->grids[indexC].edges[edgeIndex].yend;
						}
						else {
							insertPointOutX = grids->grids[indexC].edges[edgeIndex].xstart;
							insertPointOutY = grids->grids[indexC].edges[edgeIndex].ystart;
						}
						//���������䣬�����µ��߳�
						MyVector *myVectorNew = (struct MyVector *)malloc(sizeof(struct MyVector) * N);
						points[i].point[points[i].N].x = insertPointOutX;
						points[i].point[points[i].N].y = insertPointOutY;
						points[i].N++;
						if (points[i].N > MAX_REFLECTION_TIMES) {
							//������������,û�ҵ����㣬��˵���Ϊ0
							points[i].N = 0;
							points[i].isFind = false;
							break;
						}
						/*else {
							points[i].isFind = true;
						}*/
						for (int m = 0; m < N; m++) {
							//myVector[i]= (struct MyVector *)malloc(sizeof(struct MyVector));
							//��Ҫ�������д��points

							myVectorNew[m].x = insertPointOutX;
							myVectorNew[m].y = insertPointOutY;
							double angle = 2 * PIs / N * m;
							myVectorNew[m].vectorX = cos(angle);
							myVectorNew[m].vectorY = sin(angle);
							myVectorNew[m].col = int(insertPointOutX / 2);
							myVectorNew[m].row = int(insertPointOutY / 2);
							//MyVector *cuda_myVectorNew;
							//hipMalloc(&cuda_myVectorNew, sizeof(struct MyVector) * N);
							//hipMemcpy(cuda_myVectorNew, myVectorNew, sizeof(struct MyVector) * N, hipMemcpyHostToDevice);


						}
						int threadsPerBlock = 180;
						int blocksPerGrid = N / 180;
						//__syncthreads();
						judegTouchedDiffraction << <blocksPerGrid, threadsPerBlock >> > (myVectorNew, grids, diffractionPoints, point, N, i);
						//__syncthreads();
						//judgeIsTouched << <blocksPerGrid, threadsPerBlock >> > (myVectorNew, grids, points, diffractionPoints, point, N, true, i);
						//ִ�н����Ժ󽫲�������ִ��
						break;
					}
					else {
						//Severity	Code	Description	Project	File	Line	Suppression State
						//Error		kernel launch from __device__ or __global__ functions requires separate compilation mode	Demo1	D : \program\qt\R\RayTracingCUDA\raytracing3.0\Demo1\MyRayTracingCUDA.cu	574

						//��ǰ���ߵ�����
						double currentVectorX = x - insertPointOutX;
						double currentVectorY = y - insertPointOutY;
						//Ҫ��������
						//���������ǶԳ�
						// \  |  /
						//  \ | /
						//   \|/
						//------------------
						//����ǽ��ķ�����
						//grids->grids[indexC].edges[k]
						//֮ǰ���ǽ��ķ�������˳ʱ�뷽��ģ�Ҳ���ǳ��ڣ���������Ĭ������ֻ���ⲿ�����
						//��˷�������ȡ��
						double edgeVectorX = grids->grids[indexC].edges[edgeIndex].vectorX;
						double edgeVectorY = grids->grids[indexC].edges[edgeIndex].vectorY;

						if (edgeVectorX*currentVectorX + edgeVectorY * currentVectorY < 0) {
							//˵��������ǽ���ڲ������ֱ�ӽ���
							//��������
							//�н��������90������Ҫת��
							//edgeVectorX = -edgeVectorX;
							//edgeVectorY = -edgeVectorY;
							points[i].N = 0;
							points[i].isFind = false;
							break;
						}
						else {
							double temp = currentVectorX * edgeVectorX + currentVectorY * edgeVectorY;//uv
							double edgeProjectVectorX = temp * edgeVectorX;
							double edgeProjectVectorY = temp * edgeVectorY;
							//������ϣ��������ݣ�������д�����Points
							vectorX = 2 * edgeProjectVectorX - currentVectorX;
							vectorY = 2 * edgeProjectVectorY - currentVectorY;
							normalizeVector(vectorX, vectorY);//�����Ժ���Ҫ��һ��
							x = insertPointOutX;//���·������
							y = insertPointOutY;
							//if (isDiffraction) {
							//	//˵���������������,��Ҫ������д����Ӧ��λ��
							//	diffractionPoints[currentPointsIndex*N + i].point[diffractionPoints[currentPointsIndex*N + i].N].x = x;
							//	diffractionPoints[currentPointsIndex*N + i].point[diffractionPoints[currentPointsIndex*N + i].N].y = y;
							//	diffractionPoints[currentPointsIndex*N + i].N++;
							//	if (diffractionPoints[currentPointsIndex*N + i].N > MAX_REFLECTION_TIMES) {
							//		diffractionPoints[currentPointsIndex*N + i].N = 0;
							//		diffractionPoints[currentPointsIndex*N + i].isFind = false;
							//		break;
							//	}
							//}
							//else {
							points[i].point[points[i].N].x = x;
							points[i].point[points[i].N].y = y;
							points[i].N++;
							if (points[i].N > MAX_REFLECTION_TIMES) {
								//������������,û�ҵ����㣬��˵���Ϊ0
								points[i].N = 0;
								points[i].isFind = false;
								break;
							}
							//}
							row = oldRow;
							col = oldCol;
						}
						//ע�⣬ǽ��ķ�����Ϊ��λ����
						//���������ڷ������ϵ�ͶӰΪ(uv)v 
						//��������Ϊ2(uv)v-u
						//�ο�https://www.cnblogs.com/graphics/archive/2013/02/21/2920627.html


					}
					//}

				}
				else {
					x = newX;
					y = newY;
				}
			}
			else {
				//��������
				x = newX;
				y = newY;
			}
			//printf("current position:%f,%f", x, y);
			//�����newX��newY�ǽ�����������꣬��ʱ��Ҫ�жϸ����߶�������еı��Ƿ��ཻ
			//break;
			//if (grids[indexC].grids->isContains) {
			//	
			//	for (int k = 0; k < grids[indexC].grids->N; i++) {
			//		//judge weather is intersecting
			//	}

			//}
			//else {


			//	//��Ҫ�ҵ��������ĸ���
			//	//update vectors
			//	//remain direction update x and y
			//}
		}
		//vector[i].
	}
}
/*
grid_input:�洢��������Ϣ��ÿ�������д��ڱߣ���Щ��Ҳ�ṹ������
N:��ʾ�����������ȼ�����֣����N=360����ÿ���Ƕ�Ϊ1��
*/
vector<vector<double>> initCUDAInput(Grids *grids, double TX_X, double TX_Y, double RX_X, double RX_Y, int N) {
	//��Ҫ��ʼ������
	//grids->�洢����
	//������������Ϊ50*50��С������ɣ�������������С����Щ��������ȫ����Ϊ��
	//int N = 180;
	printf("Start...%d\n", sizeof(bool));
	clock_t start, finish;
	start = clock();
	struct Grids *cuda_grids;

	MyVector *myVector = (struct MyVector *)malloc(sizeof(struct MyVector) * N);
	Points *points = (struct Points *)malloc(sizeof(struct Points) * N);
	//������Ҫ����һ���ṹ�������洢����õ��Ĳ�������ط��и������ǣ����N=360������360��ray�����ܲ������䣬������ʵ�������������Ŀ����ԱȽ�С�������ô��ռ���˷���
	Points *diffractionPoints = (struct Points *)malloc(sizeof(struct Points) * N*N);
	Point *Rx = (struct Point *)malloc(sizeof(struct Point));
	Rx->x = RX_X;
	Rx->y = RX_Y;
	Rx->row = int(RX_Y / LENGTH);
	Rx->col = int(RX_X / LENGTH);

	for (int i = 0; i < N; i++) {
		//myVector[i]= (struct MyVector *)malloc(sizeof(struct MyVector));
		myVector[i].x = TX_X;
		myVector[i].y = TX_Y;
		double angle = 2 * PIs / N * i;
		myVector[i].vectorX = cos(angle);
		myVector[i].vectorY = sin(angle);
		myVector[i].col = int(TX_X / 2);
		myVector[i].row = int(TX_Y / 2);
		points[i].N = 0;
		points[i].isFind = false;
		for (int j = 0; j < N; j++) {
			diffractionPoints[i*N + j].N = 0;
			diffractionPoints[i*N + j].isFind = false;
		}
	}
	MyVector *cuda_myVector;
	Points *cuda_points;
	Point *cuda_Rx;
	Points *cuda_diffractionPoints;

	hipMalloc(&cuda_grids, sizeof(struct Grids));
	hipMalloc(&cuda_myVector, sizeof(struct MyVector) * N);
	hipMalloc(&cuda_points, sizeof(struct Points) * N);
	hipMalloc(&cuda_diffractionPoints, sizeof(struct Points) * N*N);
	hipMalloc(&cuda_Rx, sizeof(struct Point));

	hipMemcpy(cuda_grids, grids, sizeof(struct Grids), hipMemcpyHostToDevice);
	hipMemcpy(cuda_myVector, myVector, sizeof(struct MyVector) * N, hipMemcpyHostToDevice);
	hipMemcpy(cuda_points, points, sizeof(struct Points) * N, hipMemcpyHostToDevice);
	hipMemcpy(cuda_diffractionPoints, diffractionPoints, sizeof(struct Points) * N*N, hipMemcpyHostToDevice);
	hipMemcpy(cuda_Rx, Rx, sizeof(struct Point), hipMemcpyHostToDevice);
	//��ط�Ӧ�����������
	int threadsPerBlock = 90;
	int blocksPerGrid = N / 90;

	judgeIsTouched << <blocksPerGrid, threadsPerBlock >> > (cuda_myVector, cuda_grids, cuda_points, cuda_diffractionPoints, cuda_Rx, N, 0);

	hipMemcpy(points, cuda_points, sizeof(struct Points) * N, hipMemcpyDeviceToHost);
	hipMemcpy(diffractionPoints, cuda_diffractionPoints, sizeof(struct Points) * N*N, hipMemcpyDeviceToHost);
	vector<vector<double>> getResult;
	int pointsCount = 0;
	for (int i = 0; i < N; i++) {
		if (points[i].isFind) {
			vector<double> results;
			for (int j = 0; j < points[i].N; j++) {
				results.push_back(points[i].point[j].x);
				results.push_back(points[i].point[j].y);
			}
			getResult.push_back(results);
		}
		else
			if (points[i].N > 0) {
				vector<double> results;
				for (int j = 0; j < points[i].N; j++) {
					results.push_back(points[i].point[j].x);
					results.push_back(points[i].point[j].y);
				}
				//getResult.push_back(results);
				for (int m = 0; m < N; m++) {
					//qDebug() << "come to here"+ diffractionPoints[i*N + m].N;
					if (diffractionPoints[i*N + m].isFind) {
						vector<double> diffractionPo(results);
						for (int n = 0; n < diffractionPoints[i*N + m].N; n++) {
							diffractionPo.push_back(diffractionPoints[i*N + m].point[n].x);
							diffractionPo.push_back(diffractionPoints[i*N + m].point[n].y);
						}
						getResult.push_back(diffractionPo);
					}
				}
			}
	}
	//for (int i = 0; i < N; i++) {
	//	for (int j = 0; j < N; j++) {
	//		//qDebug()<<
	//		vector<double> diffractionPo;
	//		if (diffractionPoints[i*N + j].N > 0) {
	//			for (int n = 0; n < diffractionPoints[i*N + j].N; n++) {
	//				diffractionPo.push_back(diffractionPoints[i*N + j].point[n].x);
	//				diffractionPo.push_back(diffractionPoints[i*N + j].point[n].y);
	//			}
	//			getResult.push_back(diffractionPo);
	//		}

	//	}
	//}
	finish = clock();
	double duration = (double)(finish - start) / CLOCKS_PER_SEC;
	printf("%f seconds\n", duration);
	//��Ҫ����point
	hipFree(cuda_grids);
	hipFree(cuda_myVector);
	hipFree(cuda_points);
	hipFree(cuda_Rx);
	hipFree(cuda_diffractionPoints);
	free(myVector);
	return getResult;
}

vector<QString> getCUDAInformation() {
	vector<QString> output;
	hipDeviceProp_t deviceProp;
	int deviceCount;
	hipError_t hipError_t;
	hipError_t = hipGetDeviceCount(&deviceCount);
	for (int i = 0; i < deviceCount; i++)
	{
		hipError_t = hipGetDeviceProperties(&deviceProp, i);
		output.push_back(QString("%1").arg(i));
		output.push_back(QString(deviceProp.name));
		output.push_back(QString("%1").arg((int)(deviceProp.totalGlobalMem / 1024 / 1024)));
		output.push_back(QString("%1").arg((int)(deviceProp.sharedMemPerBlock / 1024)));
		output.push_back(QString("%1").arg((int)(deviceProp.regsPerBlock)));
		output.push_back(QString("%1").arg((int)deviceProp.maxThreadsPerBlock));
		output.push_back(QString("%1").arg(deviceProp.major));
		output.push_back(QString("%1").arg(deviceProp.minor));
		output.push_back(QString("%1").arg(deviceProp.multiProcessorCount));
	}
	return output;
}

//vector<vector<double>> initCUDAInput(vector<vector<vector<double>>> grid_input, double TX_X, double TX_Y, double RX_X, double RX_Y, int N) {
//	//��Ҫ��ʼ������
//	//grids->�洢����
//	//������������Ϊ50*50��С������ɣ�������������С����Щ��������ȫ����Ϊ��
//	//int N = 180;
//	printf("Start...%d\n", sizeof(bool));
//	clock_t start, finish;
//	start = clock();
//	struct Grids *grids = (struct Grids *)malloc(sizeof(struct Grids));
//	grids->width = 2;
//	grids->height = 2;
//	int i = 0;
//	for (; i < grid_input.size(); i++) {
//		if (grid_input[i].size() != 0) {
//			grids->grids[i].N = grid_input[i].size();
//			grids->grids[i].isContains = true;
//			for (int j = 0; j < grid_input[i].size(); j++) {
//				//edge
//				grids->grids[i].edges[j].xstart = grid_input[i][j][0];
//				grids->grids[i].edges[j].ystart = grid_input[i][j][1];
//				grids->grids[i].edges[j].xend = grid_input[i][j][2];
//				grids->grids[i].edges[j].yend = grid_input[i][j][3];
//				grids->grids[i].edges[j].vectorX = grid_input[i][j][4];
//				grids->grids[i].edges[j].vectorY = grid_input[i][j][5];
//			}
//		}
//		else {
//			grids->grids[i].N = 0;
//			grids->grids[i].isContains = false;
//		}
//	}
//	for (; i < ROW*COL; i++) {
//		grids->grids[i].N = 0;
//		grids->grids[i].isContains = false;
//	}
//	struct Grids *cuda_grids;
//
//	MyVector *myVector = (struct MyVector *)malloc(sizeof(struct MyVector) * N);
//	Points *points = (struct Points *)malloc(sizeof(struct Points) * N);
//	Point *Rx = (struct Point *)malloc(sizeof(struct Point));
//	Rx->x = RX_X;
//	Rx->y = RX_Y;
//	Rx->row = int(RX_Y / LENGTH);
//	Rx->col = int(RX_X / LENGTH);
//
//	for (int i = 0; i < N; i++) {
//		//myVector[i]= (struct MyVector *)malloc(sizeof(struct MyVector));
//		myVector[i].x = TX_X;
//		myVector[i].y = TX_Y;
//		double angle = 2 * PIs / N * i;
//		myVector[i].vectorX = cos(angle);
//		myVector[i].vectorY = sin(angle);
//		myVector[i].col = 0;
//		myVector[i].row = 1;
//		points[i].N = 0;
//	}
//	MyVector *cuda_myVector;
//	Points *cuda_points;
//	Point *cuda_Rx;
//	hipMalloc(&cuda_grids, sizeof(struct Grids));
//	hipMalloc(&cuda_myVector, sizeof(struct MyVector) * N);
//	hipMalloc(&cuda_points, sizeof(struct Points) * N);
//	hipMalloc(&cuda_Rx, sizeof(struct Point));
//
//	hipMemcpy(cuda_grids, grids, sizeof(struct Grids), hipMemcpyHostToDevice);
//	hipMemcpy(cuda_myVector, myVector, sizeof(struct MyVector) * N, hipMemcpyHostToDevice);
//	//hipMemcpy(cuda_points, points, sizeof(struct Points) * 60, hipMemcpyHostToDevice);
//	hipMemcpy(cuda_Rx, Rx, sizeof(struct Point), hipMemcpyHostToDevice);
//	int threadsPerBlock = 256;
//	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
//	judgeIsTouched << <blocksPerGrid, threadsPerBlock >> > (cuda_myVector, cuda_grids, cuda_points, cuda_Rx, N);
//	hipMemcpy(points, cuda_points, sizeof(struct Points) * N, hipMemcpyDeviceToHost);
//
//	vector<vector<double>> getResult;
//	for (int i = 0; i < N; i++) {
//		if (points[i].N != 0) {
//			vector<double> results;
//			for (int j = 0; j < points[i].N; j++) {
//				results.push_back(points[i].point[j].x);
//				results.push_back(points[i].point[j].y);
//				//printf("%d-%d-%d:%f,%f\n", i, j, 2 * i, points[i].point[j].x, points[i].point[j].y);
//			}
//			getResult.push_back(results);
//		}
//	}
//	finish = clock();
//	double duration = (double)(finish - start) / CLOCKS_PER_SEC;
//	printf("%f seconds\n", duration);
//	//��Ҫ����point
//	hipFree(cuda_grids);
//	hipFree(cuda_myVector);
//	hipFree(cuda_points);
//	hipFree(cuda_Rx);
//	return getResult;
//}

//int main()
//{
//	//printf("Start...%d\n", sizeof(bool));
//	vector<vector<vector<double>>> grid_input;
//
//	vector<vector<double>> grid0;
//
//	vector<double> edge0;
//	edge0.push_back(1);
//	edge0.push_back(0.5);
//	edge0.push_back(2);
//	edge0.push_back(0.5);
//	edge0.push_back(0);
//	edge0.push_back(1);
//	grid0.push_back(edge0);
//	vector<double> edge1;
//	edge1.push_back(1);
//	edge1.push_back(1.5);
//	edge1.push_back(2);
//	edge1.push_back(1.5);
//	edge1.push_back(0);
//	edge1.push_back(-1);
//	grid0.push_back(edge1);
//
//	vector<double> edge2;
//	edge2.push_back(1);
//	edge2.push_back(0.5);
//	edge2.push_back(1);
//	edge2.push_back(1.5);
//	edge2.push_back(1);
//	edge2.push_back(0);
//	grid0.push_back(edge2);
//	grid_input.push_back(grid0);
//
//	vector<vector<double>> grid1;
//	vector<double> edge3;
//	edge3.push_back(2);
//	edge3.push_back(0.5);
//	edge3.push_back(3);
//	edge3.push_back(0.5);
//	edge3.push_back(0);
//	edge3.push_back(1);
//	grid1.push_back(edge3);
//	vector<double> edge4;
//	edge4.push_back(2);
//	edge4.push_back(1.5);
//	edge4.push_back(3);
//	edge4.push_back(1.5);
//	edge4.push_back(0);
//	edge4.push_back(-1);
//	grid1.push_back(edge4);
//
//	vector<double> edge5;
//	edge5.push_back(3);
//	edge5.push_back(0.5);
//	edge5.push_back(3);
//	edge5.push_back(1.5);
//	edge5.push_back(-1);
//	edge5.push_back(0);
//	grid1.push_back(edge5);
//	grid_input.push_back(grid1);
//
//	vector<vector<double>> grid2;
//	grid_input.push_back(grid2);
//
//	vector<vector<double>> grid3;
//	vector<double> edge6;
//	edge6.push_back(6.5);
//	edge6.push_back(0.5);
//	edge6.push_back(6.5);
//	edge6.push_back(1.5);
//	edge6.push_back(1);
//	edge6.push_back(0);
//	grid3.push_back(edge6);
//	vector<double> edge7;
//	edge7.push_back(7.5);
//	edge7.push_back(0.5);
//	edge7.push_back(6.5);
//	edge7.push_back(0.5);
//	edge7.push_back(0);
//	edge7.push_back(1);
//	grid3.push_back(edge7);
//
//	vector<double> edge8;
//	edge8.push_back(7.5);
//	edge8.push_back(0.5);
//	edge8.push_back(7.5);
//	edge8.push_back(1.5);
//	edge8.push_back(-1);
//	edge8.push_back(0);
//	grid3.push_back(edge8);
//	vector<double> edge9;
//	edge9.push_back(7.5);
//	edge9.push_back(1.5);
//	edge9.push_back(6.5);
//	edge9.push_back(1.5);
//	edge9.push_back(0);
//	edge9.push_back(-1);
//	grid3.push_back(edge9);
//	grid_input.push_back(grid3);
//
//	initCUDAInput(grid_input, 1, 3, 3, 2, 180);
//	//clock_t start, finish;
//	//start = clock();
//	//struct Grids *grids = (struct Grids *)malloc(sizeof(struct Grids));
//
//	////��Ҫ��������װһ��
//	//grids->width = 2;
//	//grids->height = 2;
//
//	//grids->grids[0].N = 3;
//	//grids->grids[0].isContains = true;
//
//	//grids->grids[0].edges[0].xstart = 1;
//	//grids->grids[0].edges[0].ystart = 0.5;
//	//grids->grids[0].edges[0].xend = 2;
//	//grids->grids[0].edges[0].yend = 0.5;
//	//grids->grids[0].edges[0].vectorX = 0;
//	//grids->grids[0].edges[0].vectorY = 1;
//
//	//grids->grids[0].edges[1].xstart = 1;
//	//grids->grids[0].edges[1].ystart = 1.5;
//	//grids->grids[0].edges[1].xend = 2;
//	//grids->grids[0].edges[1].yend = 1.5;
//	//grids->grids[0].edges[1].vectorX = 0;
//	//grids->grids[0].edges[1].vectorY = -1;
//
//	//grids->grids[0].edges[2].xstart = 1;
//	//grids->grids[0].edges[2].ystart = 0.5;
//	//grids->grids[0].edges[2].xend = 1;
//	//grids->grids[0].edges[2].yend = 1.5;
//	//grids->grids[0].edges[2].vectorX = 1;
//	//grids->grids[0].edges[2].vectorY = 0;
//
//	//grids->grids[1].N = 3;
//	//grids->grids[1].isContains = true;
//
//	//grids->grids[1].edges[0].xstart = 2;
//	//grids->grids[1].edges[0].ystart = 0.5;
//	//grids->grids[1].edges[0].xend = 3;
//	//grids->grids[1].edges[0].yend = 0.5;
//	//grids->grids[1].edges[0].vectorX = 0;
//	//grids->grids[1].edges[0].vectorY = 1;
//
//	//grids->grids[1].edges[1].xstart = 2;
//	//grids->grids[1].edges[1].ystart = 1.5;
//	//grids->grids[1].edges[1].xend = 3;
//	//grids->grids[1].edges[1].yend = 1.5;
//	//grids->grids[1].edges[1].vectorX = 0;
//	//grids->grids[1].edges[1].vectorY = -1;
//
//	//grids->grids[1].edges[2].xstart = 3;
//	//grids->grids[1].edges[2].ystart = 0.5;
//	//grids->grids[1].edges[2].xend = 3;
//	//grids->grids[1].edges[2].yend = 1.5;
//	//grids->grids[1].edges[2].vectorX = -1;
//	//grids->grids[1].edges[2].vectorY = 0;
//
//	//grids->grids[2].N = 0;
//	//grids->grids[2].isContains = false;
//
//	//grids->grids[3].N = 4;
//	//grids->grids[3].isContains = true;
//
//	//grids->grids[3].edges[0].xstart = 2.5;
//	//grids->grids[3].edges[0].ystart = 2.5;
//	//grids->grids[3].edges[0].xend = 2.5;
//	//grids->grids[3].edges[0].yend = 3.5;
//	//grids->grids[3].edges[0].vectorX = 1;
//	//grids->grids[3].edges[0].vectorY = 0;
//
//	//grids->grids[3].edges[1].xstart = 3.5;
//	//grids->grids[3].edges[1].ystart = 2.5;
//	//grids->grids[3].edges[1].xend = 3.5;
//	//grids->grids[3].edges[1].yend = 3.5;
//	//grids->grids[3].edges[1].vectorX = -1;
//	//grids->grids[3].edges[1].vectorY = 0;
//
//	//grids->grids[3].edges[2].xstart = 2.5;
//	//grids->grids[3].edges[2].ystart = 2.5;
//	//grids->grids[3].edges[2].xend = 3.5;
//	//grids->grids[3].edges[2].yend = 2.5;
//	//grids->grids[3].edges[2].vectorX = 0;
//	//grids->grids[3].edges[2].vectorY = 1;
//
//	//grids->grids[3].edges[3].xstart = 2.5;
//	//grids->grids[3].edges[3].ystart = 3.5;
//	//grids->grids[3].edges[3].xend = 3.5;
//	//grids->grids[3].edges[3].yend = 3.5;
//	//grids->grids[3].edges[3].vectorX = 0;
//	//grids->grids[3].edges[3].vectorY = -1;
//
//	/*
//	�����ʼ��������ͼ��ʾ��һ��ģ�ͣ����������һ�������
//		 __ __ __ __ __ __ __ __
//		|               __ __   |
//		|     ��       |     |  |
//		|              |__ __|  |
//		|                       |
//		|      __ __ __ __      |
//		|     |           |     |
//		|     |__ __ __ __|     |
//		|__ __ __ __ __ __ __ __|
//	
//	��ֻ�ǲ���һ���ܷ���GPU���㷴�������
//	���������ܹ�ʹ�õģ�����Ϊɶ��ʹ��CPU����û�ж������
//	*/
//	/*grids.grids = new Grid[4];
//	for (int i = 0; i < 4; i++) {
//		grids.grids[i].N = 4;
//		grids.grids[i].isContains = true;
//		grids.grids[i].edges = new Edge[4];
//
//		for (int j = 0; j < 4; j++) {
//			grids.grids[i].edges[j].xstart = 0;
//			grids.grids[i].edges[j].ystart = 0 + i;
//			grids.grids[i].edges[j].xend = 0;
//			grids.grids[i].edges[j].yend = 1 + j;
//			grids.grids[i].edges[j].vectorX = 1;
//			grids.grids[i].edges[j].vectorY = 1;
//		}
//	}*/
//
//	//printf("grids's size%d\n", sizeof(struct Grids));
//	//struct Grids *cuda_grids;
//	//int N = 180;
//	//MyVector *myVector = (struct MyVector *)malloc(sizeof(struct MyVector) * N);
//	//Points *points = (struct Points *)malloc(sizeof(struct Points) * N);
//	//Point *Rx = (struct Point *)malloc(sizeof(struct Point));
//	//Rx->x = 3;
//	//Rx->y = 3;
//
//	//for (int i = 0; i < N; i++) {
//	//	//myVector[i]= (struct MyVector *)malloc(sizeof(struct MyVector));
//	//	myVector[i].x = 1;
//	//	myVector[i].y = 3;
//	//	double angle = 2 * PIs / N * i;
//	//	myVector[i].vectorX = cos(angle);
//	//	myVector[i].vectorY = sin(angle);
//	//	myVector[i].col = 0;
//	//	myVector[i].row = 1;
//	//	points[i].N = 0;
//	//}
//	//MyVector *cuda_myVector;
//	//Points *cuda_points;
//	//Point *cuda_Rx;
//	//hipMalloc(&cuda_grids, sizeof(struct Grids));
//	//hipMalloc(&cuda_myVector, sizeof(struct MyVector) * N);
//	//hipMalloc(&cuda_points, sizeof(struct Points) * N);
//	//hipMalloc(&cuda_Rx, sizeof(struct Point));
//
//	//hipMemcpy(cuda_grids, grids, sizeof(struct Grids), hipMemcpyHostToDevice);
//	//hipMemcpy(cuda_myVector, myVector, sizeof(struct MyVector) * N, hipMemcpyHostToDevice);
//	////hipMemcpy(cuda_points, points, sizeof(struct Points) * 60, hipMemcpyHostToDevice);
//	//hipMemcpy(cuda_Rx, Rx, sizeof(struct Point), hipMemcpyHostToDevice);
//
//
//	//int threadsPerBlock = 256;
//	//int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
//	//judgeIsTouched <<<blocksPerGrid, threadsPerBlock >>> (cuda_myVector, cuda_grids, cuda_points, cuda_Rx, N);
//	//hipMemcpy(points, cuda_points, sizeof(struct Points) * N, hipMemcpyDeviceToHost);
//
//
//	//for (int i = 0; i < N; i++) {
//	//	for (int j = 0; j < points[i].N; j++) {
//	//		printf("%d-%d-%d:%f,%f\n",i,j, 2*i,points[i].point[j].x, points[i].point[j].y);
//	//	}
//	//}
//	//finish = clock();
//	//double duration = (double)(finish - start) / CLOCKS_PER_SEC;
//	//printf("%f seconds\n", duration);
//	//hipFree(cuda_grids);
//	//hipFree(cuda_myVector);
//	//hipFree(cuda_points);
//	//hipFree(cuda_Rx);
//
//
//	//for (int i = 0; i < 4; i++) {
//	//	for (int j = 0; j < 4; j++) {
//	//		printf("%d\n", grids.grids[i].edges[j].xstart);
//	//		//std::cout << grids.grids[i].edges[j].xstart << "," << grids.grids[i].edges[j].ystart << "," << grids.grids[i].edges[j].xend << "," << grids.grids[i].edges[j].yend << std::endl;
//	//	}
//	//}
//	//hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//	//if (cudaStatus != hipSuccess) {
//	//	fprintf(stderr, "addWithCuda failed!");
//	//	return 1;
//	//}
//
//	//printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//	//	c[0], c[1], c[2], c[3], c[4]);
//
//	//// hipDeviceReset must be called before exiting in order for profiling and
//	//// tracing tools such as Nsight and Visual Profiler to show complete traces.
//	//cudaStatus = hipDeviceReset();
//	//if (cudaStatus != hipSuccess) {
//	//	fprintf(stderr, "hipDeviceReset failed!");
//	//	return 1;
//	//}
//
//	return 0;
//}
//
